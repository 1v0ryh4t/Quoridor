#include "hip/hip_runtime.h"
 //#include <wb.h>
#include <stdio.h> 
extern "C"{
	#include "aiMinMax.h"
	#include "scan.h" 
};

 #define block_size 512 //@@ you can change this
  
 /*#define wbchecestmt) \
 do { \
 cudaerror_t err = stmt; \
 if (err != cudasuccess) { \
 wblog(error, "failed to run stmt ", #stmt); \
 return -1; \
 } \
 } while (0)*/

 __global__ void fixup(float *input, float *aux, int len) {
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * block_size;
    if (blockIdx.x) {
       if (start + t < len)
         input[start + t] += aux[blockIdx.x - 1];//these are also add lines
       if (start + block_size + t < len)
         input[start + block_size + t] += aux[blockIdx.x - 1];//these are also add lines
    }
 }

 __global__ void scan(float *input, float *output, float *aux, int len) {
 // load a segment of the input vector into shared memory
   __shared__ float scan_array[block_size << 1];
   unsigned int t = threadIdx.x, start = 2 * blockIdx.x * block_size;
   if (start + t < len)
     scan_array[t] = input[start + t];
   else
     scan_array[t] = 0;
   if (start + block_size + t < len)
      scan_array[block_size + t] = input[start + block_size + t];
   else
     scan_array[block_size + t] = 0;
   __syncthreads();
 
   // reduction
   int stride;
   for (stride = 1; stride <= block_size; stride <<= 1) {
      int index = (t + 1) * stride * 2 - 1;
      if (index < 2 * block_size)
        scan_array[index] += scan_array[index - stride];//this line is the add line
      __syncthreads();
   }

   // post reduction
   for (stride = block_size >> 1; stride; stride >>= 1) {
      int index = (t + 1) * stride * 2 - 1;
      if (index + stride < 2 * block_size)
        scan_array[index + stride] += scan_array[index];//this line is the add line
      __syncthreads();
   }

   if (start + t < len)
     output[start + t] = scan_array[t];
   if (start + block_size + t < len)
     output[start + block_size + t] = scan_array[block_size + t];

   if (aux && t == 0)
      aux[blockIdx.x] = scan_array[2 * block_size - 1];
   }

int takeVal(Value val) {
   //wbarg_t args;
   float *hostinput; // the input 1d list
   float *hostoutput; // the output list
   float *deviceinput;
   float *deviceoutput;
   float *deviceauxarray, *deviceauxscannedarray;
   int numelements=val->size; // number of elements in the list

   //args = wbarg_read(argc, argv);

   //wbtime_start(generic, "importing data and creating memory on host");
   
   float hostarr[numelements];
   for(int i=0;i<numelements;i++){
 	hostarr[i] = val->v;
	printf("hostarr[%d] = %f",i,hostarr[i]);
   }
   hostinput = hostarr;

 //(float *)wbimport(wbarg_getinputfile(args, 0), &numelements);//make this call a method from sequential ai that generates a list
   hipHostAlloc(&hostoutput, numelements * sizeof(float), hipHostMallocDefault);
   //wbtime_stop(generic, "importing data and creating memory on host");

   //wblog(trace, "the number of input elements in the input is ", numelements);

   //wbtime_start(gpu, "allocating gpu memory.");

   hipMalloc((void **)&deviceinput, numelements * sizeof(float));
   hipMalloc((void **)&deviceoutput, numelements * sizeof(float));

   // xxx the size is fixed for ease of implementation.
   hipMalloc(&deviceauxarray, (block_size << 1) * sizeof(float));
   hipMalloc(&deviceauxscannedarray, (block_size << 1) * sizeof(float));
   //wbtime_stop(gpu, "allocating gpu memory.");

   //wbtime_start(gpu, "clearing output memory.");
   hipMemset(deviceoutput, 0, numelements * sizeof(float));
   //wbtime_stop(gpu, "clearing output memory.");

    //wbtime_start(gpu, "copying input memory to the gpu.");
    hipMemcpy(deviceinput, hostinput, numelements * sizeof(float),hipMemcpyHostToDevice);
    //wbtime_stop(gpu, "copying input memory to the gpu.");

    //@@ initialize the grid and block dimensions here
    int numblocks = ceil((float)numelements / (block_size << 1));
    dim3 dimgrid(numblocks, 1, 1);
    dim3 dimblock(block_size, 1, 1);
    //wblog(trace, "the number of blocks is ", numblocks);

    //wbtime_start(compute, "performing cuda computation");
    //@@ modify this to complete the functionality of the scan
    //@@ on the deivce
    scan<<<dimgrid, dimblock>>>(deviceinput, deviceoutput, deviceauxarray, numelements);
    hipDeviceSynchronize();
    scan<<<dim3(1, 1, 1), dimblock>>>(deviceauxarray, deviceauxscannedarray, NULL, block_size << 1);
    hipDeviceSynchronize();
    fixup<<<dimgrid, dimblock>>>(deviceoutput, deviceauxscannedarray, numelements);

    hipDeviceSynchronize();
    //wbtime_stop(compute, "performing cuda computation");

    //wbtime_start(copy, "copying output memory to the cpu");
    hipMemcpy(hostoutput, deviceoutput, numelements * sizeof(float),
    hipMemcpyDeviceToHost);
    //wbtime_stop(copy, "copying output memory to the cpu");

    //wbtime_start(gpu, "freeing gpu memory");
    hipFree(deviceinput);
    hipFree(deviceoutput);
    hipFree(deviceauxarray);
    hipFree(deviceauxscannedarray);
    //wbtime_stop(gpu, "freeing gpu memory");

    //wbsolution(args, hostoutput, numelements);
    int ret = *((int*)(&hostinput));
    free(hostinput);
    hipHostFree(hostoutput);

    return ret;
 }
