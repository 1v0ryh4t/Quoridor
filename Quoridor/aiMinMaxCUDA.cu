#include "hip/hip_runtime.h"
//#include <wb.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
extern "C"{
	#include "setup.h"
	#include "play.h"
	#include "globals.h"
	#include "aiMinMax.h"
}

#define thread_size 175
#define block_size 512
#define size 9
#define max_moves 175 //max leegs with a little extra

//how scancode relates: just do 1 pass down to calc terminals then 1 pass to calc min or max 
// in first pass should just stride loop
// each stride should be maxLegals^treedepth (passes are different stride locations) 
//if invalid option, set value to unfavorable
//look at inefficient work kernel (just slap something together)

//leegs is boardstate, depth is treedepth, minning and maxing are whether we are running min or max, finalZero, finalOne and finalOri are returned row, col and ori
__global__ void treeGen(int row, int col, char* color, char** walltrack, int depth, int minOrMax, int* array){

	//1 is white, 0 is black

	
	int t = blockIdx.x*blockDim.x+threadIdx.x;
        //determine whether a legal has kids (pawns at winning or depth = 0) this for loop is here because inefficient scan kernel
        for(int stride = 1;stride<=threadIdx.x;stride *= depth){
                //go through legalactions here
                __syncthreads();
                
                //how to assign min and max layers?
                //what to return? how to return top move? go back up tree minning and maxing
                //how to go back up tree? reduction kernel type thing

		//too much of this kernel relies on sequential method calls. I don't know if it will work
                
                if(t<max_moves){
                        float * leegs = getLegalMoves(size,walltrack,color,row,col);
			int decider = (int)(*(leegs+t)+0);
                        	if((strcmp(color,"black")==0 && row == 0) || (strcmp(color,"white")==0 && row == size-1)){
                        		array[t] = 1000;
                        	}
				int minpathw=size*size, minpathb=size*size;
                        	minpathsMinMax(size, walltrack, &minpathw, &minpathb);
                        	if(minpathw<minpathb && strcmp("white",color)==0){//these will trigger early
                        		array[t] = 500;
                        	}
                        	if(minpathb<minpathw && color==0){
                        		array[t] = 500;
                        	}
                        	if((decider<0) && minOrMax==1){
                        		array[t] = -999;
                        	}
                        	if((decider<0) && minOrMax==0){
                        		array[t] = 999;
                        	}
                        	if((!decider<0) && t<6){
                        		int newrow = (int)(*(leegs+t)+0);
                        		int newcol = (int)(*(leegs+t)+1);
                        		leegs = getLegalMoves(size,walltrack,color,newrow,newcol);
                        	}
                        	if((!decider<0) && t<87  && t>5){
                        		walltrack[(int)(*(leegs+t)+0)][(int)(*(leegs+t)+1)] = 'H';
                        		leegs = getLegalMoves(size,walltrack,color,row,col);
                        	}
                        	if((!leegs[t]<0) && t<172 && t>86){
                        		walltrack[(int)(*(leegs+t)+0)][(int)(*(leegs+t)+1)] = 'V';
                        		leegs = getLegalMoves(size,walltrack,color,row,col);
                        	}
                        	
                        }
	
                              // reduction
			   int stride;
			   for (stride = 1; stride <= block_size; stride <<= 1) {
			      int index = (t + 1) * stride * 2 - 1;
			      if (index < 2 * block_size){
			      	if(stride%(depth)==0){
			        //compare instead of add
			        array[index] = min(array[index],array[index-stride]);//how to get 173 vals? like this?
			      __syncthreads();
			      }
			      else{
			      	array[index] = max(array[index],array[index-stride]);//how to get 173 vals?
			      __syncthreads();
			      }
			   }
			} 
}
}

